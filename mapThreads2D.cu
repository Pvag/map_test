
#include <hip/hip_runtime.h>
#include <stdio.h>

#define SEP_LINE_LENGTH 20

typedef struct gridTopology
{
  dim3 blockSize;
  dim3 gridSize;
} gridTopology;

typedef struct pixelCoords
{
  int x, y;
} pixelCoords;

gridTopology initGridTopology2D(int r, int c);
void gridDataReport(gridTopology t, int nRows, int nCols);
void printLineOf(char c);
__device__ int validThread(gridTopology t);
__global__ void testInvalidThreads(int r, int c);

int main(int argc, char *argv[])
{
  int nRows = 10, nCols = 10;
  gridTopology t = initGridTopology2D(nRows, nCols);
  // gridDataReport(t, nRows, nCols);

  testInvalidThreads<<<t.gridSize, t.blockSize>>>(nRows, nCols);

  hipDeviceSynchronize(); // flush the printf of threads !

  // cudaDeviceReset();

  return EXIT_SUCCESS;
}

gridTopology initGridTopology2D(int nRows, int nCols)
{
  hipDeviceProp_t deviceProp;
  int dev = 0;
  hipGetDeviceProperties(&deviceProp, dev);
  int maxGridSizeX = deviceProp.maxGridSize[0];
  int maxGridSizeY = deviceProp.maxGridSize[1];
  int maxThreadsX = deviceProp.maxThreadsDim[0];
  int maxThreadsY = deviceProp.maxThreadsDim[1];

  gridTopology t;

  int blockSizeX = sqrt(deviceProp.maxThreadsPerBlock);
  if (blockSizeX > maxThreadsX) perror("! Error");
  int blockSizeY = blockSizeX;
  if (blockSizeY > maxThreadsY) { }
  int nBlocksX = (nCols + blockSizeX - 1) / blockSizeX;
  if (nBlocksX > maxGridSizeX) { }
  int nBlocksY = (nRows + blockSizeY - 1) / blockSizeY;
  if (nBlocksY > maxGridSizeY) { }

  t.blockSize = dim3(blockSizeX, blockSizeY, 0);
  t.gridSize = dim3(nBlocksX, nBlocksY, 0);
  return t;
}

void gridDataReport(gridTopology t, int nRows, int nCols)
{
  printLineOf('*');
  printf("Sizes of blocks: x:%d, y:%d\n", t.blockSize.x,
      t.blockSize.y);
  printf("Sizes of grid: x:%d, y:%d\n", t.gridSize.x,
      t.gridSize.y);
  printf("nRows: %d, nCols: %d; total pixels: %d\n",
      nRows, nCols, nRows * nCols);
  printf("number of threads: %d\n",
      t.blockSize.x * t.blockSize.y //
      * t.gridSize.x * t.gridSize.y);
  printf("number of threads - 1 block in x - 1 block in y: %d\n",
      t.blockSize.x * t.blockSize.y //
      * (t.gridSize.x - 1) * (t.gridSize.y - 1) );
  printLineOf('*');
}

__device__ int validThread(pixelCoords p)
{
  // Checks if the thread should compute or not,
  // according to its position in the CUDA grid,
  // with respect to the original 2D matrix size - e.g. an image.
  return (p.x > -1 && p.y > -1) ? 1 : 0;
}

__device__ pixelCoords computeThread2DCoordinates(int r, int c)
{
  // Each function that uses this function MUST check
  // that both pixelX and pixelY are != -1 .

  int pixelX = blockIdx.x * blockDim.x + threadIdx.x;
  int pixelY = blockIdx.y * blockDim.y + threadIdx.y;
  // NOTE: row and column of the matrix have +1 w.r.t. thread
  //       x and y coordinates !
  if (pixelX >= c) pixelX = -1; // this thread is out of bounds
  if (pixelY >= r) pixelY = -1; // this thread is out of bounds

  pixelCoords p = {pixelX, pixelY};
  return p;
}

__global__ void testInvalidThreads(int r, int c)
{
  printf("ok\n");
//  pixelCoords p = computeThread2DCoordinates(r, c);
//  if (!validThread(p))
//    printf("Thread x:%d y:%d has nothing to do here.\n",
//      p.x, p.y);
//  else printf("ciao\n");
}

void printLineOf(const char c)
{
  int i;
  for (i = 0; i < SEP_LINE_LENGTH; i++)
  {
    printf(" %c", c);
  }
  printf("\n");
}
